#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

__global__ void addKernel(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    const int N = 1024;
    float *d_a, *d_b, *d_c;
    
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));
    
    addKernel<<<4, 256>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasDestroy(handle);
    
    hipFree(d_a);
    hipFree(d_b); 
    hipFree(d_c);
    return 0;
}
